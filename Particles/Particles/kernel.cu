#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <SFML/Graphics.hpp>
#include <SFML/OpenGL.hpp>
#include <stdio.h>
#include <stdlib.h>

#define EARTH_KG 100

double * h_pos;
double * h_vel;
double * h_acc;
double * h_mass;

double * d_pos;
double * d_vel;
double * d_acc;
double* d_mass;

int numParticles = 1000;


sf::Shader pointProgram;
sf::Shader niceProgram;

sf::RenderTexture buffer;
sf::RenderTexture finalBuffer;

sf::RenderWindow window(sf::VideoMode(1024, 1024), "SFML works!");

inline double random(const double& max, const double& min = 0)
{
	return ((double)static_cast <float> (rand()) / static_cast <float> (RAND_MAX)) * max + min;
}

inline void genParticles()
{
	int particlesGenerated = 0;
	float xDim = 150000;
	float yDim = 150000;
	float zDim = 150000;

	for (int i = 0; i < numParticles; i++)
	{
		int index = particlesGenerated * 3;
		h_pos[index] = random(xDim, -xDim) + xDim / 2;
		h_pos[index + 1] = random(yDim, -yDim) + yDim / 2;
		h_pos[index + 2] = random(zDim, -zDim) + zDim / 2;
		h_vel[index] = random(50, -25);
		h_vel[index + 1] = random(50, -25);
		h_vel[index + 2] = random(12.5, -6);
		h_acc[index] = 0;
		h_acc[index + 1] = 0;
		h_acc[index + 2] = 0;
		h_mass[particlesGenerated] = EARTH_KG;
		particlesGenerated++;
	}
}

inline void drawParticles()
{
	buffer.clear(sf::Color(0, 0, 0, 0));
	finalBuffer.clear(sf::Color(0, 0, 0, 0));

	sf::Sprite mySprite(buffer.getTexture());
	pointProgram.setParameter("texture", buffer.getTexture());

	finalBuffer.draw(mySprite, &pointProgram);

	mySprite.setTexture(finalBuffer.getTexture());
	niceProgram.setParameter("texture", finalBuffer.getTexture());

	window.draw(mySprite, &niceProgram);
}

int main()
{
	int pointsPerParticleVec = 3;
	size_t size = sizeof(double) * pointsPerParticleVec * numParticles;

	h_pos = (double*)malloc(size);
	h_vel = (double*)malloc(size);
	h_acc = (double*)malloc(size);
	h_mass = (double*)malloc(size / pointsPerParticleVec);

	d_pos = NULL;
	d_vel = NULL;
	d_acc = NULL;
	hipError_t err = hipSuccess;
	//allocate space on GPU
	err = hipMalloc((void **)&d_pos, size);
	err = hipMalloc((void **)&d_vel, size);
	err = hipMalloc((void **)&d_acc, size);
	err = hipMalloc((void **)&d_mass, size / 3);

	genParticles();

	//copy from cpu to GPU
	size_t size = sizeof(double) * 3 * numParticles;
	hipError_t err = hipSuccess;
	err = hipMemcpy(d_pos, h_pos, size, hipMemcpyHostToDevice);
	err = hipMemcpy(d_vel, h_vel, size, hipMemcpyHostToDevice);
	err = hipMemcpy(d_acc, h_acc, size, hipMemcpyHostToDevice);
	err = hipMemcpy(d_mass, h_mass, size / 3, hipMemcpyHostToDevice);






	pointProgram.loadFromFile("pointVertex.glsl", "pointFragment.glsl");

	niceProgram.loadFromFile("vertex.glsl", "fragment.glsl");

	while (window.isOpen())
	{
		sf::Event event;
		while (window.pollEvent(event))
		{
			if (event.type == sf::Event::Closed)
				window.close();
		}
		//gpuupdate
		window.clear(sf::Color(0, 0, 0, 0));
		
		drawParticles();



		window.display();
	}

	return 0;
}


