

#include <hip/hip_runtime.h>
#include ""
#include <iostream>
#include <stdio.h>
#include <chrono>

#include "ParticleRenderer.hpp"
#include "ParticleSystem.hpp"

#include "math.cuh"
#include "noise\noise.h"



int main(int argc, char** argv)
{
	srand(time(NULL));

	ParticleRenderer ren;

	glutInit(&argc, argv);

	ren.initGL();
	ren.initSystem();
	ren.begin();

	
}