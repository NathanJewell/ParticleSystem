#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <SFML/Graphics.hpp>
#include <SFML/OpenGL.hpp>
#include <stdio.h>
#include <stdlib.h>

#define EARTH_KG 100

double * h_pos;
double * h_vel;
double * h_acc;
double * h_mass;

double * d_pos;
double * d_vel;
double * d_acc;
double* d_mass;

int numParticles = 1000;


sf::Shader pointProgram;
sf::Shader niceProgram;

sf::RenderTexture buffer;
sf::RenderTexture finalBuffer;

sf::RenderWindow window(sf::VideoMode(1024, 1024), "SFML works!");

inline double random(const double& max, const double& min = 0)
{
	return ((double)static_cast <float> (rand()) / static_cast <float> (RAND_MAX)) * max + min;
}

inline void genParticles()
{
	int particlesGenerated = 0;
	float xDim = 150000;
	float yDim = 150000;
	float zDim = 150000;

	for (int i = 0; i < numParticles; i++)
	{
		int index = particlesGenerated * 3;
		h_pos[index] = random(xDim, -xDim) + xDim / 2;
		h_pos[index + 1] = random(yDim, -yDim) + yDim / 2;
		h_pos[index + 2] = random(zDim, -zDim) + zDim / 2;
		h_vel[index] = random(50, -25);
		h_vel[index + 1] = random(50, -25);
		h_vel[index + 2] = random(12.5, -6);
		h_acc[index] = 0;
		h_acc[index + 1] = 0;
		h_acc[index + 2] = 0;
		h_mass[particlesGenerated] = EARTH_KG;
		particlesGenerated++;
	}
}

inline void drawParticles()
{
	buffer.clear(sf::Color(0, 0, 0, 0));
	finalBuffer.clear(sf::Color(0, 0, 0, 0));
	buffer.resetGLStates();


	glClearColor(0.0, 0.0, 0.0, 0.0);
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

	glColor4f(1.0, 0.0, 0.0, .4);
	glBegin(GL_POINTS);
	for (int i = 0; i < numParticles; i++)
	{
		int index = i * 3;
		glVertex3f(h_pos[index]/100, h_pos[index + 1]/100, h_pos[index + 2]/100);
	}
	glEnd();

	buffer.pushGLStates();

	sf::Sprite mySprite(buffer.getTexture());
	pointProgram.setParameter("texture", buffer.getTexture());

	buffer.popGLStates();

	finalBuffer.draw(mySprite, &pointProgram);

	mySprite.setTexture(finalBuffer.getTexture());
	niceProgram.setParameter("texture", finalBuffer.getTexture());

	window.draw(mySprite, &niceProgram);
}

int main()
{
	int pointsPerParticleVec = 3;
	size_t size = sizeof(double) * pointsPerParticleVec * numParticles;

	h_pos = (double*)malloc(size);
	h_vel = (double*)malloc(size);
	h_acc = (double*)malloc(size);
	h_mass = (double*)malloc(size / pointsPerParticleVec);

	d_pos = NULL;
	d_vel = NULL;
	d_acc = NULL;
	hipError_t err = hipSuccess;
	//allocate space on GPU
	err = hipMalloc((void **)&d_pos, size);
	err = hipMalloc((void **)&d_vel, size);
	err = hipMalloc((void **)&d_acc, size);
	err = hipMalloc((void **)&d_mass, size / 3);

	genParticles();

	//copy from cpu to GPU


	err = hipMemcpy(d_pos, h_pos, size, hipMemcpyHostToDevice);
	err = hipMemcpy(d_vel, h_vel, size, hipMemcpyHostToDevice);
	err = hipMemcpy(d_acc, h_acc, size, hipMemcpyHostToDevice);
	err = hipMemcpy(d_mass, h_mass, size / 3, hipMemcpyHostToDevice);






	pointProgram.loadFromFile("pointVertex.glsl", "pointFragment.glsl");

	niceProgram.loadFromFile("vertex.glsl", "fragment.glsl");

	int width = window.getSize().x;
	int height = window.getSize().y;
	glViewport(0, 0, 1024, 1024);
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glOrtho(-width / 2, width / 2, -height / 2, height / 2, -1000000000, 1000000000);
	glMatrixMode(GL_MODELVIEW);
	glLoadIdentity();

	//glMatrixMode(GL_PROJECTION);

	//glOrtho(0.0, 1.0, 0.0, 1.0, -1.0, 1.0);

	glLoadIdentity();

	//glEnable(GL_DEPTH_TEST);
	glClearColor(0.0f, 0.0f, 0.0f, 0.0f);
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
	sf::Event event;
	while (window.pollEvent(event))
	{
		if (event.type == sf::Event::Closed)
		{
			window.close();
		}
				
		//gpuupdate
		window.clear(sf::Color(0, 0, 0, 0));
		
		drawParticles();



		window.display();
	}

	return 0;
}


